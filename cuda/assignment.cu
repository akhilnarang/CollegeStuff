#include "hip/hip_runtime.h"

#include <stdio.h>

#define MAX 10

int size = MAX * sizeof(int);

__global__ void cudasum(int *a, int *b, int *c) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  c[i] = a[i] + b[i];
}

int main() {
  int a[MAX], b[MAX], c[MAX];
  int *da, *db, *dc;
  int i;
  for (i = 0; i < MAX; i++) {
    a[i] = b[i] = i;
  }

  hipMalloc(&da, size);
  hipMalloc(&db, size);
  hipMalloc(&dc, size);
  hipMemcpy(da, a, size, hipMemcpyHostToDevice);
  hipMemcpy(db, b, size, hipMemcpyHostToDevice);
  hipMemcpy(dc, c, size, hipMemcpyHostToDevice);
  cudasum<<<1, 10>>>(da, db, dc);
  hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);
  for (i = 0; i < MAX; i++) {
    printf("c[%d] = %d\n", i, c[i]);
  }
  return 0;
}